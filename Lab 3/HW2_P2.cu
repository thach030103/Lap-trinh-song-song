
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)\
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", error,\
				hipGetErrorString(error));\
		exit(EXIT_FAILURE);\
	}\
}
#define TILE_WIDTH 32
struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

__global__ void matrix_multiplication_kernel1(float* A, float* B, float* C, int m, int n, int k)
{
	//TODO
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if (col < k && row < m)
    {
        for (int i = 0; i < n; i++)
        {
            sum += A[row * n + i] * B[i * k + col];
        }
        C[row * k + col] = sum;
    }
}

__global__ void matrix_multiplication_kernel2(float* A, float* B, float* C, int m, int n, int k)
{
	__shared__ float s_A[TILE_WIDTH][TILE_WIDTH];
	__shared__ float s_B[TILE_WIDTH][TILE_WIDTH];

	//TODO
    int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;

    float Pvalue = 0.0f;  

    for (int t = 0; t < (n - 1) / TILE_WIDTH + 1; t++) 
    {
        if (Row < m && t * TILE_WIDTH + threadIdx.x < n)
            s_A[threadIdx.y][threadIdx.x] = A[Row * n + t * TILE_WIDTH + threadIdx.x];
        else
            s_A[threadIdx.y][threadIdx.x] = 0.0f;

        if (Col < k && t * TILE_WIDTH + threadIdx.y < n)
            s_B[threadIdx.y][threadIdx.x] = B[(t * TILE_WIDTH + threadIdx.y) * k + Col];
        else
            s_B[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; ++i)
            Pvalue += s_A[threadIdx.y][i] * s_B[i][threadIdx.x];

        __syncthreads();
    }

    if (Row < m && Col < k)
        C[Row * k + Col] = Pvalue;
}

void matrix_multiplication(float* A, float* B, float* C, int m, int n, int k,
    bool useDevice = false, dim3 blockSize = dim3(1),int kernelType=1)
{
    GpuTimer timer;
    timer.Start();
    if (useDevice == false)
    {
        // TODO
        for (int row = 0; row < m; ++row) 
        {
            for (int col = 0; col < k; ++col) 
            {
                float sum = 0;
                for (int i = 0; i < n; ++i) 
                {
                    sum += A[row * n + i] * B[i * k + col];
                }
                C[row * k + col] = sum;
            }
        }
    }
    else // Use device
    {
        // TODO: Allocate device memories
        float* d_A, * d_B, * d_C;
        hipMalloc(&d_A, m * n * sizeof(float));
        hipMalloc(&d_B, n * k * sizeof(float));
        hipMalloc(&d_C, k * m * sizeof(float));

        // TODO: Copy data to device memories
        hipMemcpy(d_A, A, m * n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, n * k * sizeof(float), hipMemcpyHostToDevice);
        
        //dim3 gridSize(1); // TODO: Compute gridSize
        dim3 gridSize((k - 1) / blockSize.x + 1, (m - 1) / blockSize.y + 1);
        
		if (kernelType == 1)
			matrix_multiplication_kernel1<<<gridSize, blockSize>>>(d_A, d_B, d_C, m, n, k);
		else if (kernelType == 2)
			matrix_multiplication_kernel2<<<gridSize, blockSize>>>(d_A, d_B, d_C, m, n, k);

        // TODO: Copy result from device memory
        hipMemcpy(C, d_C, m * k * sizeof(float), hipMemcpyDeviceToHost);

        // TODO: Free device memories
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
		
		printf("Grid size: %d * %d, block size: %d * %d\n", 
			gridSize.x,gridSize.y, blockSize.x,blockSize.y);

    }
    timer.Stop();
    float time = timer.Elapsed();
    printf("Processing time (%s): %f ms\n",
        useDevice == true ? "use device" : "use host", time);
}

float checkCorrectness(float * a1, float* a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)	
		err += abs(a1[i] - a2[i]);
	err /= n;
	return err;
}

void printDeviceInfo()
{
	hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
    printf("****************************\n\n");

}
int main(int argc, char** argv)
{
	printDeviceInfo();
	
	//Declare variables
    float* h_A; // The A matrix
    float* h_B; // The B matrix
    float* h_C; // The output C matrix
    float* correct_C; // The output C matrix

    int m;    // number of rows in the matrix A
    int n; // number of columns in the matrix A, number of rows in the matrix B
    int k; // number of columns in the matrix B

    m = (1 << 10);
    n = (1 << 9);
    k = (1 << 10);

    // Set up input data
    h_A = (float*)malloc(m * n * sizeof(float));
    h_B = (float*)malloc(n * k * sizeof(float));
    h_C = (float*)malloc(m * k * sizeof(float));
    correct_C = (float*)malloc(m * k * sizeof(float));

    for (int i = 0; i < m; i++)
        for (int j = 0;j < n;j++)
            h_A[i*n+j] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
 
    for (int i = 0; i < n; i++)
        for (int j = 0;j < k;j++)
            h_B[i*k+j] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);


    // Add vectors (on host)
    matrix_multiplication(h_A,h_B,correct_C,m,n,k);
	printf("\n");

	dim3 blockSize(32, 32); // Default
	if (argc == 3)
	{
		blockSize.x = atoi(argv[1]);
		blockSize.y = atoi(argv[2]);
	} 
    // Add in1 & in2 on device
	printf("Basic Matrix Multiplication:\n");
    matrix_multiplication(h_A, h_B, h_C, m, n, k, true,blockSize,1);
	float err = checkCorrectness(h_C, correct_C,m*k);
	printf("Error between device result and host result: %f\n\n", err);

	printf("Shared memory Matrix Multiplication:\n");
    matrix_multiplication(h_A, h_B, h_C, m, n, k, true,blockSize,2);
	err = checkCorrectness(h_C, correct_C,m*k);
	printf("Error between device result and host result: %f\n\n", err);	
	
    free(h_A);
    free(h_B);
    free(h_C);
    free(correct_C);

    return 0;
}
