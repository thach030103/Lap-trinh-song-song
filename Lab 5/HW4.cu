
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(1);\
    }\
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

// Sequential Radix Sort
void sortByHost(const uint32_t * in, int n,
                uint32_t * out)
{
    int * bits = (int *)malloc(n * sizeof(int));
    int * nOnesBefore = (int *)malloc(n * sizeof(int));

    uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
    uint32_t * originalSrc = src; // To free memory later
    memcpy(src, in, n * sizeof(uint32_t));
    uint32_t * dst = out;

    // Loop from LSB (Least Significant Bit) to MSB (Most Significant Bit)
	// In each loop, sort elements according to the current bit from src to dst 
	// (using STABLE counting sort)
    for (int bitIdx = 0; bitIdx < sizeof(uint32_t) * 8; bitIdx++)
    {
        // Extract bits
        for (int i = 0; i < n; i++)
            bits[i] = (src[i] >> bitIdx) & 1;

        // Compute nOnesBefore
        nOnesBefore[0] = 0;
        for (int i = 1; i < n; i++)
            nOnesBefore[i] = nOnesBefore[i-1] + bits[i-1];

        // Compute rank and write to dst
        int nZeros = n - nOnesBefore[n-1] - bits[n-1];
        for (int i = 0; i < n; i++)
        {
            int rank;
            if (bits[i] == 0)
                rank = i - nOnesBefore[i];
            else
                rank = nZeros + nOnesBefore[i];
            dst[rank] = src[i];
        }

        // Swap src and dst
        uint32_t * temp = src;
        src = dst;
        dst = temp;
    }

    // Does out array contain results?
    memcpy(out, src, n * sizeof(uint32_t));

    // Free memory
    free(originalSrc);
    free(bits);
    free(nOnesBefore);
}

__device__ int bCount = 0;  
__device__ int bCount1 = 0;

__global__ void extractBits(const uint32_t *in, int *bits, int n, int bitIdx) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        bits[i] = (in[i] >> bitIdx) & 1;
    }
}

__global__ void exclusiveScan(int *in, int n, int *out, int *blkSums)
{
    // TODO
	// 1. Each block loads data from GMEM to SMEM
	extern __shared__ int s_data[];
	int i1 = blockIdx.x * 2 * blockDim.x + threadIdx.x;
	int i2 = i1 + blockDim.x;
	if (i1 < n)
    {
		s_data[threadIdx.x] = in[i1];
    }
    else
    {       
        s_data[threadIdx.x] = 0;
    }
	if (i2 < n)
    {
		s_data[threadIdx.x + blockDim.x] = in[i2];
    }
    else
    {
        s_data[threadIdx.x] = 0;
    }
	__syncthreads();

	// 2. Each block does scan with data on SMEM
	// 2.1. Reduction phase
	for (int stride = 1; stride < 2 * blockDim.x; stride *= 2)
	{
		int s_dataIdx = (threadIdx.x + 1) * 2 * stride - 1; // To avoid warp divergence
		if (s_dataIdx < 2 * blockDim.x)
			s_data[s_dataIdx] += s_data[s_dataIdx - stride];
		__syncthreads();
	}
	// 2.2. Post-reduction phase
	for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
	{
		int s_dataIdx = (threadIdx.x + 1) * 2 * stride - 1 + stride; // Wow
		if (s_dataIdx < 2 * blockDim.x)
			s_data[s_dataIdx] += s_data[s_dataIdx - stride];
		__syncthreads();
	}

	// 3. Each block writes results from SMEM to GMEM
	if (i1 < n)
		out[i1] = s_data[threadIdx.x];
	if (i2 < n)
		out[i2] = s_data[threadIdx.x + blockDim.x];

	if (blkSums != NULL && threadIdx.x == 0)
		blkSums[blockIdx.x] = s_data[2 * blockDim.x - 1];

    if (threadIdx.x == 0)
    {
        s_data[0] = atomicAdd(&bCount,1);
    }
    __syncthreads();

    int bid = s_data[0];

    if (threadIdx.x == 0)
    {
        blkSums[bid] = s_data[2 * blockDim.x - 1];
        if (bid > 0){
            while (atomicAdd(&bCount1, 0) < bid) {}
            s_data[blockDim.x * 2] = blkSums[bid - 1];
            blkSums[bid] += s_data[blockDim.x * 2];
            __threadfence();
        }
        atomicAdd(&bCount1, 1);
    }
    __syncthreads();

}

__global__ void addPrevBlkSum(int * blkSumsScan, int * blkScans, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockDim.x;
    if (i < n)
        blkScans[i] += blkSumsScan[blockIdx.x];
}

// Compute rank and write to out
__global__ void scatter(const uint32_t *in, uint32_t *out, int *bits, int *nOnesBefore, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        int nZeros = n - nOnesBefore[n - 1] - bits[n - 1]; // Tính số lượng bit 0
        int rank;

        if (bits[i] == 0) {
            rank = i - nOnesBefore[i]; // Vị trí cho bit 0
        } else {
            rank = nZeros + nOnesBefore[i]; // Vị trí cho bit 1
        }
        out[rank] = in[i];
    }
}

// Parallel Radix Sort
void sortByDevice(const uint32_t * in, int n, uint32_t * out, int blockSize)
{
    // TODO
    // 1. Khởi tạo bộ nhớ thiết bị
    uint32_t *d_in, *d_out;
    int *d_bits, *d_nOnesBefore, *d_blkSums, *d_blkSumsScan;

    CHECK(hipMalloc(&d_in, n * sizeof(uint32_t)));
    CHECK(hipMalloc(&d_out, n * sizeof(uint32_t)));
    CHECK(hipMalloc(&d_bits, n * sizeof(int)));
    CHECK(hipMalloc(&d_nOnesBefore, n * sizeof(int)));

    int gridSize((n - 1) / blockSize + 1);
    CHECK(hipMalloc(&d_blkSums, gridSize * sizeof(int)));
    CHECK(hipMalloc(&d_blkSumsScan, gridSize * sizeof(int)));

    CHECK(hipMemcpy(d_in, in, n * sizeof(uint32_t), hipMemcpyHostToDevice));

    // 2. Vòng lặp qua từng bit
    for (int bitIdx = 0; bitIdx <  sizeof(uint32_t) * 8; bitIdx++)
    {
        // 2.1. Rút trích bit
        extractBits<<<gridSize, blockSize>>>(d_in, d_bits, n, bitIdx);
        CHECK(hipDeviceSynchronize());

        // 2.2. Exclusive scan
        exclusiveScan<<<gridSize, blockSize, 2 * blockSize * sizeof(int)>>>(d_bits, n, d_nOnesBefore, d_blkSums);
        CHECK(hipDeviceSynchronize());

        //Reset lại biến đếm
        CHECK(hipMemcpyToSymbol(HIP_SYMBOL(bCount), 0, sizeof(int)));
        CHECK(hipMemcpyToSymbol(HIP_SYMBOL(bCount1), 0, sizeof(int)));

        addPrevBlkSum<<<gridSize, blockSize>>>(d_blkSums, d_nOnesBefore, n);
        CHECK(hipDeviceSynchronize());

        // 2.3. Phân phối (scatter)
        scatter<<<gridSize, blockSize>>>(d_in, d_out, d_bits, d_nOnesBefore, n);
        CHECK(hipDeviceSynchronize());

        uint32_t *temp = d_in;
        d_in = d_out;
        d_out = temp;
    }

    CHECK(hipMemcpy(out, d_in, n * sizeof(uint32_t), hipMemcpyDeviceToHost));


    // 5. Giải phóng bộ nhớ thiết bị
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_bits);
    hipFree(d_nOnesBefore);
    hipFree(d_blkSums);
    hipFree(d_blkSumsScan);

}
// Radix Sort
void sort(const uint32_t * in, int n, 
        uint32_t * out, 
        bool useDevice=false, int blockSize=1)
{
    GpuTimer timer; 
    timer.Start();

    if (useDevice == false)
    {
    	printf("\nRadix Sort by host\n");
        sortByHost(in, n, out);
    }
    else // use device
    {
    	printf("\nRadix Sort by device\n");
        sortByDevice(in, n, out, blockSize);
    }

    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
}

void printDeviceInfo()
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
    printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
    printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
    printf("****************************\n");
}

void checkCorrectness(uint32_t * out, uint32_t * correctOut, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (out[i] != correctOut[i])
        {
            printf("INCORRECT :(\n");
            return;
        }
    }
    printf("CORRECT :)\n");
}

void printArray(uint32_t * a, int n)
{
    for (int i = 0; i < n; i++)
        printf("%i ", a[i]);
    printf("\n");
}

int main(int argc, char ** argv)
{
    // PRINT OUT DEVICE INFO
    printDeviceInfo();

    // SET UP INPUT SIZE
    //int n = 50; // For test by eye
    int n = (1 << 24) + 1;
    printf("\nInput size: %d\n", n);

    // ALLOCATE MEMORIES
    size_t bytes = n * sizeof(uint32_t);
    uint32_t * in = (uint32_t *)malloc(bytes);
    uint32_t * out = (uint32_t *)malloc(bytes); // Device result
    uint32_t * correctOut = (uint32_t *)malloc(bytes); // Host result

    // SET UP INPUT DATA
    for (int i = 0; i < n; i++)
    {
        //in[i] = rand() % 255; // For test by eye
        in[i] = rand();
    }
    //printArray(in, n); // For test by eye

    // DETERMINE BLOCK SIZE
    int blockSize = 512; // Default 
    if (argc == 2)
        blockSize = atoi(argv[1]);

    // SORT BY HOST
    sort(in, n, correctOut);
    //printArray(correctOut, n); // For test by eye
    
    // SORT BY DEVICE
    sort(in, n, out, true, blockSize);
    //printArray(out, n); // For test by eye
    checkCorrectness(out, correctOut, n);

    // FREE MEMORIES
    free(in);
    free(out);
    free(correctOut);
    
    return EXIT_SUCCESS;
}
